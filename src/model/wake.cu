#include "hip/hip_runtime.h"
/*
 * Wakes in the simulation environment
 *
 * Author: Roice (LUO Bing)
 * Date: 2016-03-08 create this file (RAOS)
 */

#include <stdio.h>
#include "model/robot.h"
#include "model/wake_rotor.h"
#include "model/error_cuda.h"
#include "model/plume.h" // for wake-induced velocity computation of plume puffs

#define PI 3.14159265358979323846

// storing cuda device properties, for the sheduling of parallel computing
static hipDeviceProp_t prop;

// data
VortexMarker_t* wake_markers; // on-host array containing all of the markers, for convenience mem copying with GPU
VortexMarker_t* dev_wake_markers; // on-device array ...
VortexMarker_t* dev_wake_markers_mediate; // on-device array ...
int* idx_end_marker_fila; // on-host array containing index of endpoint of all vortex filaments, for parallel computing
int* dev_idx_end_marker_fila; // on-device array ...

// vel += biot savert induction from segement a-b to position p
__device__ float3 biot_savart_induction(VortexMarker_t a, VortexMarker_t b, float3 p, float3 vel)
{
    float3 ap, bp, ab, ba, ind;
    float dbnrm_ab, dbnrm_ap, dbnrm_bp; // |vector|^2
    float cos_apab, cos_babp, db_sin_apab; // cos(ap,ab) cos(ba,bp) sin(ap,ab)^2
    float db_h, scale; // h^2, |ind|

    // if marker a or b is at p, pass
    if ( a.pos[0]==p.x && a.pos[1]==p.y && a.pos[2]==p.z ||
         b.pos[0]==p.x && b.pos[1]==p.y && b.pos[2]==p.z ||
         a.pos[0]==b.pos[0] && a.pos[1]==b.pos[1] && a.pos[2]==b.pos[2]) {
        return vel;
    }

    // get vectors AP, BP and AB & BA  [12 FLOPS]
    //  AP [3 FLOPS] 
    ap.x = p.x - a.pos[0];
    ap.y = p.y - a.pos[1];
    ap.z = p.z - a.pos[2];
    /* // __fsub_ru() costs equal to simply minus
    ap.x = __fsub_ru(p.x, a.pos[0]);
    ap.y = __fsub_ru(p.y, a.pos[1]);
    ap.z = __fsub_ru(p.z, a.pos[2]);
    */
    //  BP [3 FLOPS]
    bp.x = p.x - b.pos[0];
    bp.y = p.y - b.pos[1];
    bp.z = p.z - b.pos[2];
    /*
    bp.x = __fsub_ru(p.x, b.pos[0]);
    bp.y = __fsub_ru(p.y, b.pos[1]);
    bp.z = __fsub_ru(p.z, b.pos[2]);
    */
    //  AB [3 FLOPS]
    ab.x = b.pos[0] - a.pos[0];
    ab.y = b.pos[1] - a.pos[1];
    ab.z = b.pos[2] - a.pos[2];
    /*
    ab.x = __fsub_ru(b.pos[0], a.pos[0]);
    ab.y = __fsub_ru(b.pos[1], a.pos[1]);
    ab.z = __fsub_ru(b.pos[2], a.pos[2]);
    */
    //  BA [3 FLOPS]
    ba.x = a.pos[0] - b.pos[0];
    ba.y = a.pos[1] - b.pos[1];
    ba.z = a.pos[2] - b.pos[2];
    /*
    ba.x = __fsub_ru(a.pos[0], b.pos[0]);
    ba.y = __fsub_ru(a.pos[1], b.pos[1]);
    ba.z = __fsub_ru(a.pos[2], b.pos[2]);
    */

    // cos(ap-ab) and cos(ba-bp) [31 FLOPS]
    dbnrm_ab = ab.x*ab.x + ab.y*ab.y + ab.z*ab.z; // [5 FLOPS]
    dbnrm_ap = ap.x*ap.x + ap.y*ap.y + ap.z*ap.z; // [5 FLOPS]
    dbnrm_bp = bp.x*bp.x + bp.y*bp.y + bp.z*bp.z; // [5 FLOPS] 
    /* // using __powf(x,y) to calculate squares costs more than simply multiply
    dbnrm_ab = __powf(ab.x,2) + __powf(ab.y,2) + __powf(ab.z,2);
    dbnrm_ap = __powf(ap.x,2) + __powf(ap.y,2) + __powf(ap.z,2);
    dbnrm_bp = __powf(bp.x,2) + __powf(bp.y,2) + __powf(bp.z,2);
    */
    cos_apab = (ap.x*ab.x + ap.y*ab.y + ap.z*ab.z)*__frsqrt_rn(dbnrm_ap*dbnrm_ab); // [8 FLOPS]
    cos_babp = (ba.x*bp.x + ba.y*bp.y + ba.z*bp.z)*__frsqrt_rn(dbnrm_ab*dbnrm_bp); // [8 FLOPS]

    // h, perpendicular distance from P to AB [3 FLOPS]
    db_sin_apab = 1-cos_apab*cos_apab; //[2 FLOPS]
    if (db_sin_apab <= 0.0f) // sometimes |cos_apab| will be slightly larger than 1.0f due to computation err
        db_h = 0.0f;
    else {
        db_h = dbnrm_ap*db_sin_apab; // [1 FLOPS]
    }

    // strength
    scale = a.Gamma * db_h * __frsqrt_rn(__powf(a.r,4)+db_h*db_h) * (cos_apab+cos_babp) / (4*PI);
 
    // get induced velocity
    ind.x = ap.y*bp.z - ap.z*bp.y;
    ind.y = ap.z*bp.x - ap.x*bp.z;
    ind.z = ap.x*bp.y - ap.y*bp.x;
    scale = scale * __frsqrt_rn(ind.x*ind.x + ind.y*ind.y + ind.z*ind.z);
    ind.x = scale * ind.x;
    ind.y = scale * ind.y;
    ind.z = scale * ind.z;

    // add induced velocity [3 FLOPS]
    vel.x += ind.x;
    vel.y += ind.y;
    vel.z += ind.z;

    return vel;
}

__device__ float3 tile_calculation_vel_markers
(float3 pos, float3 vel, int tile, int tile_size,int row_sgmts, VortexMarker_t* markers, int* index_end, int num_fila, int num_markers)
{
    int i; // the i-th colum marker in this tile
    int idx_m; // the idx_m -th colum marker of whole N markers
    int idx_fila; // index of fila
    int shared_mem_offset = (tile%row_sgmts)*tile_size;

    bool isend; // the marker to be calculated is an end point or not

    extern __shared__ VortexMarker_t tile_markers[];
 
    // if this tile is not the first tile, we will use the last tile's end
    if (tile > 0)
    {
        isend = false;
        for (idx_fila = 0; idx_fila < num_fila; idx_fila++) {
            if (tile*tile_size-1 == index_end[idx_fila]) {
                isend = true;
                break;
            }
        }
        if (isend == false && tile*tile_size < num_markers)
            vel = biot_savart_induction(markers[tile*tile_size-1], tile_markers[shared_mem_offset], pos, vel); 
    }

    for (i = 0; i < tile_size-1; i++)
    {// compule all colum markers in this tile
        idx_m = tile * tile_size + i;
        if (idx_m >= num_markers-1)
            break;
        else
        {
            isend = false;
            // check whether this marker is end point or not
            for (idx_fila = 0; idx_fila < num_fila; idx_fila++) {
                if (idx_m == index_end[idx_fila]) {
                    isend = true;
                    break;
                }
            }
            if (isend == false) // if this marker is not an end point
                vel = biot_savart_induction(tile_markers[shared_mem_offset+i], tile_markers[shared_mem_offset+i+1], pos, vel);
        }
    }

    return vel;
}

/* calculate velocities of markers, running on GPU
Note:
    This routine uses one-dimensional blocks and threads
    row_sgms is the number of segments to parallelly compute a row
    blockDim must equal to tile_size*row_sgmts
    gridDim == (num_markers + tile_size - 1) / tile_size
 */
__global__ void CalculateVelofMarkers(VortexMarker_t* markers, int* idx_end, int num_fila, int num_markers, int tile_size, int row_sgmts)
{
    extern __shared__ VortexMarker_t tile_markers[];

    int idx, i;
    float3 pos; // position of vortex marker to calculate velocity in this thread
    float3 vel = {0.0f, 0.0f, 0.0f}; // velocity of this marker
    int row = threadIdx.x%tile_size + blockIdx.x*tile_size; // get row number (the index of marker to be evaluated)

    // get the marker of which the velocity to be calculated
    if (row < num_markers) {
        pos.x = markers[row].pos[0];
        pos.y = markers[row].pos[1];
        pos.z = markers[row].pos[2];
    }

    // compute each tile
    i = 0;
    while(true) // every thread can enter this loop
    {
        idx = threadIdx.x + blockDim.x*i;
        if (idx < num_markers)
            tile_markers[threadIdx.x] = markers[idx]; // copy markers to shared mem for tile calculation 
        __syncthreads(); // make sure the shared mem has been loaded
        if (row < num_markers)
            vel = tile_calculation_vel_markers(pos, vel, idx/tile_size, tile_size, row_sgmts, markers, idx_end, num_fila, num_markers);
        __syncthreads(); // make sure every thread has done the calc of this tile
        
        if (blockDim.x*(i+1) < num_markers)
            i++;
        else
            break;
    }

    // save the result of this segment to the end of the shared mem
    //  at present the shared memory is of no use for computation, so it can be used to temporarily store the segment results
    if (row < num_markers) {
        tile_markers[threadIdx.x].vel[0] = vel.x;
        tile_markers[threadIdx.x].vel[1] = vel.y;
        tile_markers[threadIdx.x].vel[2] = vel.z;
    }
    __syncthreads(); // make sure the segment results are stored
 
    if (threadIdx.x < tile_size && row < num_markers) {
        // sum the velocities computed by multiple (num of row_sgms) threads
        vel.x = 0.0f; vel.y = 0.0f; vel.z = 0.0f;
        for (i = 0; i < row_sgmts; i++) {
            vel.x += tile_markers[threadIdx.x%tile_size+i*tile_size].vel[0];
            vel.y += tile_markers[threadIdx.x%tile_size+i*tile_size].vel[1];
            vel.z += tile_markers[threadIdx.x%tile_size+i*tile_size].vel[2];
        }
        // Save the result in global memory for the integration step.
        markers[row].vel[0] = vel.x;
        markers[row].vel[1] = vel.y;
        markers[row].vel[2] = vel.z; 
    }
}

/* calculate new positions of markers, running on GPU 
    Forward Euler  
 */
__global__ void CalculatePosofMarkersFE(VortexMarker_t* markers, int num_markers, float dt)
{
    VortexMarker_t mkr;
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if (tid < num_markers) {
        mkr = markers[tid];

        for (int i = 0; i < 3; i++)
            mkr.pos[i] += mkr.vel[i]*dt;
    }
    if (tid < num_markers)
        markers[tid] = mkr;
}

/* average velocity of two sequence of markers
   the number of markers should be the same
   This routine is used for the correction step of predictor-corrector integration
   scheme of FVM, which involves the average computation of previous velocity and
   predicted velocity

   the vel of array markers and markers_temp will be averaged and stored in markers
 */
__global__ void AverageVelofMarkers(VortexMarker_t* markers, VortexMarker_t* markers_temp, int num_markers)
{
    VortexMarker_t mkr, mkr_temp;
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if (tid < num_markers) {
        mkr = markers[tid];
        mkr_temp = markers_temp[tid];

        for (int i = 0; i < 3; i++)
            mkr.vel[i] = 0.5*(mkr.vel[i] + mkr_temp.vel[i]);
    }
    if (tid < num_markers)
        markers[tid] = mkr;   
}

/* calculate vortex core radius of markers */
__global__ void CalculateVtxCoreofMarkers(VortexMarker_t* markers, int num_fila, int num_markers, float dt)
{
    VortexMarker_t mkr;
    float t;
    int length;
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if (tid < num_markers) {
        mkr = markers[tid];

        // calculate life time of this marker (vortex segment)
        length = num_markers / num_fila;
        t = (float)(tid % length) / (float)length;
        /* r_c(t) = sqrt( r_init^2 + 4*alpha*delta*nu*t )
           Here we assume r_init is 0.005 m
           alpha is Lamb's constant which is 1.25643
           delta is is a function of vortex Reynolds number, delta > 1 (laminar when delta = 1)
           here we assume delta = 8
           nu is viscous constant of air at 25 degree temperature: 0.01834
        */
        mkr.r = __fsqrt_ru(0.005f*0.005f + 4.0f*1.25643f*8*0.01834*t);
    }
    if (tid < num_markers)
        markers[tid] = mkr;
}


/* update all of the rotor wakes in the environment
 * this routine will traverse all of the robot instances
 * and update all of the Lagrangian markers for all robots */
void WakesUpdate(std::vector<Robot*>* robots, const char* integration_scheme)
{
    int idx_robot, idx_rotor, idx_blade;
    int num_blade = 0;
    int addr_cp_markers = 0; // index for copy marker states from rotor wake to wake_markers

/* Step 1: update velocity & position of markers */
 
    // Phase 1: collect all vortex markers to a memory, for GPU computing
    //  the markers are placed contiguously, fila to fila
    for(idx_robot = 0; idx_robot < robots->size(); idx_robot++) {
        for (idx_rotor = 0; idx_rotor < robots->at(idx_robot)->wakes.size(); idx_rotor++) {
            for (idx_blade = 0; idx_blade < robots->at(idx_robot)->wakes.at(idx_rotor)->rotor_state.frame.n_blades; idx_blade++) {
                std::copy(robots->at(idx_robot)->wakes.at(idx_rotor)->wake_state[idx_blade]->begin(),
                    robots->at(idx_robot)->wakes.at(idx_rotor)->wake_state[idx_blade]->end(),
                    &wake_markers[addr_cp_markers]);
                addr_cp_markers += robots->at(idx_robot)->wakes.at(idx_rotor)->wake_state[idx_blade]->size();
                idx_end_marker_fila[num_blade] = addr_cp_markers-1; // the address of the last element, hence -1
                num_blade++;
            }
        }
    }// traversed all rotor wakes and got total number of markers

    // Phase 2: copy array wake_markers & idx_wake_markers to GPU's dev_wake_markers
    HANDLE_ERROR( hipMemcpy(dev_wake_markers, wake_markers, 
                addr_cp_markers*sizeof(VortexMarker_t), hipMemcpyHostToDevice) );
    HANDLE_ERROR( hipMemcpy(dev_idx_end_marker_fila, idx_end_marker_fila, 
                num_blade*sizeof(int), hipMemcpyHostToDevice) );
    
    // Phase 3: parallel computing
    //  determine threads per block and blocks number, at present addr_cp_markers contains total num of markers
    int p, q, threads, blocks;
    p = prop.warpSize; // tile size
    q = 4; // number of segments of a row
    threads = std::min(p*q, prop.maxThreadsPerBlock);  

    //  launch gpu computing
    //   the end address of shared memory contains the end element of 'last' tile for 'next' tile computation
    hipError_t err; 
    
    //  <1> calculate velocity of Lagrangian markers, n-1 state
    blocks = (addr_cp_markers + p - 1) / p;
    //  Note: here omitted checks for max number of blocks, since in RAO problem the vortex markers
    //        rarely exceeds 65535*threads.
    CalculateVelofMarkers<<<blocks, threads, (threads)*sizeof(VortexMarker_t)>>>(dev_wake_markers, dev_idx_end_marker_fila, num_blade, addr_cp_markers, p, q);
    err = hipGetLastError();
    if (err != hipSuccess)
        printf("Error: %s\n", hipGetErrorString(err));
    
    //  <2> calculate position of Lagrangian markers, predict
    blocks = (addr_cp_markers + threads -1)/threads;
    CalculatePosofMarkersFE<<<blocks, threads>>>(dev_wake_markers, addr_cp_markers, 0.001);
    err = hipGetLastError();
    if (err != hipSuccess) 
        printf("Error: %s\n", hipGetErrorString(err));
    
    if (strcmp(integration_scheme, "PC") == 0) // predictor-corrector
    {
    //  <3> calculate mediate velocity of predicted positions for corrector
    HANDLE_ERROR( hipMemcpy(dev_wake_markers_mediate, dev_wake_markers, 
                addr_cp_markers*sizeof(VortexMarker_t), hipMemcpyDeviceToDevice) );
    blocks = (addr_cp_markers + p - 1) / p;
    CalculateVelofMarkers<<<blocks, threads, (threads)*sizeof(VortexMarker_t)>>>(dev_wake_markers_mediate, dev_idx_end_marker_fila, num_blade, addr_cp_markers, p, q);
    err = hipGetLastError();
    if (err != hipSuccess)
        printf("Error: %s\n", hipGetErrorString(err));
    //  <4> calculate final position of markers, correct
    blocks = (addr_cp_markers + threads -1)/threads;
    AverageVelofMarkers<<<blocks, threads>>>(dev_wake_markers, dev_wake_markers_mediate, addr_cp_markers);
    err = hipGetLastError();
    if (err != hipSuccess) 
        printf("Error: %s\n", hipGetErrorString(err));
    CalculatePosofMarkersFE<<<blocks, threads>>>(dev_wake_markers, addr_cp_markers, 0.001);
    err = hipGetLastError();
    if (err != hipSuccess) 
        printf("Error: %s\n", hipGetErrorString(err));
    }
    //  <5> calculate vortex core growth
    //    the argument num_blade, num_markers and dt are used to compute lifetime
    blocks = (addr_cp_markers + threads -1)/threads;
    CalculateVtxCoreofMarkers<<<blocks, threads>>>(dev_wake_markers, num_blade, addr_cp_markers, 0.001);
    err = hipGetLastError();
    if (err != hipSuccess) 
        printf("Error: %s\n", hipGetErrorString(err));

    // Phase 4: retrieve data dev_wake_markers from GPU to wake_markers
    HANDLE_ERROR( hipMemcpy(wake_markers, dev_wake_markers,
                addr_cp_markers*sizeof(VortexMarker_t), hipMemcpyDeviceToHost) );

    // Phase 5: GPU computed, distribute the memory to every rotor wake states
    addr_cp_markers = 0;
    for(idx_robot = 0; idx_robot < robots->size(); idx_robot++) {
        for (idx_rotor = 0; idx_rotor < robots->at(idx_robot)->wakes.size(); idx_rotor++) {
            for (idx_blade = 0; idx_blade < robots->at(idx_robot)->wakes.at(idx_rotor)->rotor_state.frame.n_blades; idx_blade++) {
                std::copy(&wake_markers[addr_cp_markers],
                    &wake_markers[addr_cp_markers+robots->at(idx_robot)->wakes.at(idx_rotor)->wake_state[idx_blade]->size()],
                robots->at(idx_robot)->wakes.at(idx_rotor)->wake_state[idx_blade]->data());
                addr_cp_markers += robots->at(idx_robot)->wakes.at(idx_rotor)->wake_state[idx_blade]->size();
            }
        }
    }


/* Step 2: maintain markers of wakes */
    for(idx_robot = 0; idx_robot < robots->size(); idx_robot++)
    {
        for (idx_rotor = 0; idx_rotor < robots->at(idx_robot)->wakes.size(); idx_rotor++) {
            robots->at(idx_robot)->wakes.at(idx_rotor)->maintain();
        }
    }
}

/* init parallelization of wakes computation */
void WakesInit(std::vector<Robot*>* robots)
{
    /* print the properties of all the graphic cards this machine has */ 
    int count; // number of devices
    HANDLE_ERROR( hipGetDeviceCount(&count) );
    for (int i = 0; i < count; i++) {// print out info of all graphic cards
        HANDLE_ERROR( hipGetDeviceProperties(&prop, i) );
        printf("======== Card %d ========\n", i+1);
        printf("Graphic card name: %s\n", prop.name);
        printf("Compute Capability: %d.%d\n", prop.major, prop.minor);
        printf("Total global memory: %ld MByte\n", prop.totalGlobalMem/1024/1024);
        printf("Total constant memoty: %ld kByte\n", prop.totalConstMem/1024);
        printf("Multiprocessor count: %d\n", prop.multiProcessorCount);
        printf("Shared mem per mp: %ld\n", prop.sharedMemPerBlock);
        printf("Registers per mp: %d\n", prop.regsPerBlock);
        printf("Threads in warp: %d\n", prop.warpSize);
        printf("Max threads per block: %d\n", prop.maxThreadsPerBlock);
        printf("Max thread dimensions: (%d, %d, %d)\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
        printf("Max grid dimensions: (%d, %d, %d)\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
        printf("\n");
    }
    if (count > 1) {// multiple graphic cards
        printf("Warning: Multiple graphic cards have been found on this machine. Please modify the function WakeInit in the file src/model/wake.cu to choose the most appropriate card.\n");
        exit(EXIT_FAILURE); // force the user to choose which card to use
    }
    else if (count <= 0) {// no graphic card found
        printf("Error: No graphic cards have been found on this machine. Please run this program on the machine with NVIDIA graphic cards.\n");
        exit(EXIT_FAILURE);
    }

    /* Init GPU computation */
    // Note: the number of rotor wakes & vortex filaments are fixed since simulation starts

    // traverse all rotor wakes and got total max number of markers and fila, for allocating mem
    int max_num_markers = 0;
    int max_num_fila = 0;
    for(int idx_robot = 0; idx_robot < robots->size(); idx_robot++) {// traverse all robots
        for (int idx_rotor = 0; idx_rotor < robots->at(idx_robot)->wakes.size(); idx_rotor++) {// traverse all rotors
            for (int idx_blade = 0; idx_blade < robots->at(idx_robot)->wakes.at(idx_rotor)->rotor_state.frame.n_blades; idx_blade++)
            {// traverse all blades
                max_num_fila++;
                max_num_markers += robots->at(idx_robot)->wakes.at(idx_rotor)->max_markers;
            }
        }
    }

    // allocate a page-locked host memory containing all of the marker states
    HANDLE_ERROR( hipHostAlloc((void**)&wake_markers, 
        max_num_markers*sizeof(*wake_markers), hipHostMallocDefault) );
    // allocate device memory as big as the host's
    HANDLE_ERROR( hipMalloc((void**)&dev_wake_markers, 
        max_num_markers*sizeof(*dev_wake_markers)) );
    HANDLE_ERROR( hipMalloc((void**)&dev_wake_markers_mediate, 
        max_num_markers*sizeof(*dev_wake_markers_mediate)) );

    // allocate host memory containing the index of wake_markers
    HANDLE_ERROR( hipHostAlloc((void**)&idx_end_marker_fila,
        max_num_fila*sizeof(*idx_end_marker_fila), hipHostMallocDefault) );
    // allocate device memory containing the indexes
    HANDLE_ERROR( hipMalloc((void**)&dev_idx_end_marker_fila,
        max_num_fila*sizeof(*dev_idx_end_marker_fila)) ); 
}

/* close GPU computation */
void WakesFinish(void)
{
    // free device memory
    HANDLE_ERROR( hipFree(dev_idx_end_marker_fila) );
    HANDLE_ERROR( hipFree(dev_wake_markers) );
    // free host memory
    HANDLE_ERROR( hipHostFree(idx_end_marker_fila) );
    HANDLE_ERROR( hipHostFree(wake_markers) );
}

/*************** Calculate Induced Velocity at Plume puffs ***************/
// these functions can be called after WakesInit

__global__ void CalculateIndVelatPlumePuffs(VortexMarker_t* markers, int* idx_end, int num_fila, int num_markers, FilaState_t* plume, int num_puffs)
{
    float3 pos; // position of plume puff to calculate velocity in this thread
    float3 vel = {0.0f, 0.0f, 0.0f}; // velocity of this marker
    int tid = threadIdx.x + blockIdx.x * blockDim.x; // ID of this thread
    int idx_fila, i;
    bool isend; // the marker to be calculated is an end point or not

    // get the plume fila (puff) which the velocity to be calculated
    if (tid < num_puffs) 
    {
        pos.x = plume[tid].pos[0];
        pos.y = plume[tid].pos[1];
        pos.z = plume[tid].pos[2];

        for (i = 0; i < num_markers-1; i++) // every thread can enter this function
        {// traverse every vortex segments
            isend = false;
            for (idx_fila = 0; idx_fila < num_fila; idx_fila++) {
                if (i == idx_end[idx_fila]) {
                    isend = true;
                    break;
                }
            }
            if (isend == false)
                vel = biot_savart_induction(markers[i], markers[i+1], pos, vel);
        }
    }

    // save velocity
    if (tid < num_puffs) {
        plume[tid].vel[0] = vel.x;
        plume[tid].vel[1] = vel.y;
        plume[tid].vel[2] = vel.z;
    }
}

FilaState_t* plume_puffs; // on-host ...
FilaState_t* dev_plume_puffs; // on-device array containing the states of plume puffs

void WakesIndVelatPlumePuffsUpdate(std::vector<Robot*>* robots, std::vector<FilaState_t>* plume)
{
    int idx_robot, idx_rotor, idx_blade;
    int addr_cp_markers = 0, num_blade = 0;
    // Step 1: collect all vortex markers & puffs to one memory, respectively, for GPU computing
    //  the markers are placed contiguously, fila to fila
    for(idx_robot = 0; idx_robot < robots->size(); idx_robot++) {
        for (idx_rotor = 0; idx_rotor < robots->at(idx_robot)->wakes.size(); idx_rotor++) {
            for (idx_blade = 0; idx_blade < robots->at(idx_robot)->wakes.at(idx_rotor)->rotor_state.frame.n_blades; idx_blade++) {
                std::copy(robots->at(idx_robot)->wakes.at(idx_rotor)->wake_state[idx_blade]->begin(),
                    robots->at(idx_robot)->wakes.at(idx_rotor)->wake_state[idx_blade]->end(),
                    &wake_markers[addr_cp_markers]);
                addr_cp_markers += robots->at(idx_robot)->wakes.at(idx_rotor)->wake_state[idx_blade]->size();
                idx_end_marker_fila[num_blade] = addr_cp_markers-1; // the address of the last element, hence -1
                num_blade++;
            }
        }
    }// traversed all rotor wakes and got total number of markers
    //  collect puffs to a memory
    std::copy(plume->begin(), plume->end(), &plume_puffs[0]);

    // Step 2: copy array wake_markers, idx_wake_markers, plume_puffs to GPU's version
    HANDLE_ERROR( hipMemcpy(dev_wake_markers, wake_markers, 
                addr_cp_markers*sizeof(VortexMarker_t), hipMemcpyHostToDevice) );
    HANDLE_ERROR( hipMemcpy(dev_idx_end_marker_fila, idx_end_marker_fila, 
                num_blade*sizeof(int), hipMemcpyHostToDevice) );
    HANDLE_ERROR( hipMemcpy(dev_plume_puffs, plume_puffs,
                plume->size()*sizeof(FilaState_t), hipMemcpyHostToDevice) );

    // Step 3: compute induced velocity of plume puffs
    //  determine threads per block and blocks number, at present addr_cp_markers contains total num of markers
    int p, q, threads, blocks;
    p = prop.warpSize; // tile size
    q = 4; // number of segments of a row
    threads = std::min(p*q, prop.maxThreadsPerBlock);  
    //  launch gpu computing
    hipError_t err; 
    //  <1> calculate velocity of Lagrangian markers, n-1 state
    blocks = (addr_cp_markers + threads -1) / threads;
    //  Note: here omitted checks for max number of blocks, since in RAO problem the vortex markers
    //        rarely exceeds 65535*threads.
    CalculateIndVelatPlumePuffs<<<blocks, threads>>>(dev_wake_markers, dev_idx_end_marker_fila, num_blade, addr_cp_markers, dev_plume_puffs, plume->size());
    err = hipGetLastError();
    if (err != hipSuccess)
        printf("Error: %s\n", hipGetErrorString(err));

    // Step 4: distribute puffs to host and give back
    HANDLE_ERROR( hipMemcpy(plume_puffs, dev_plume_puffs,
                plume->size()*sizeof(FilaState_t), hipMemcpyDeviceToHost) );
    std::copy(&plume_puffs[0], &plume_puffs[plume->size()], plume->data());
}

void WakesIndVelatPlumePuffsInit(std::vector<Robot*>* robots, std::vector<FilaState_t>* plume)
{
    // allocate a page-locked host memory containing all of the plume puffs states
    HANDLE_ERROR( hipHostAlloc((void**)&plume_puffs, 
        MAX_NUM_PUFFS*sizeof(*plume_puffs), hipHostMallocDefault) );
    // allocate device memory as big as the host's
    HANDLE_ERROR( hipMalloc((void**)&dev_plume_puffs, 
        MAX_NUM_PUFFS*sizeof(*dev_plume_puffs)) );
}

/* End of file wake.cu */
