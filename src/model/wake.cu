#include "hip/hip_runtime.h"
/*
 * Wakes in the simulation environment
 *
 * Author: Roice (LUO Bing)
 * Date: 2016-03-08 create this file (RAOS)
 */

#include <stdio.h>
#include "model/robot.h"
#include "model/wake_rotor.h"
#include "model/error_cuda.h"

#define PI 3.14159265358979323846

// storing cuda device properties, for the sheduling of parallel computing
static hipDeviceProp_t prop;

// data
VortexMarker_t* wake_markers; // on-host array containing all of the markers, for convenience mem copying with GPU
VortexMarker_t* dev_wake_markers; // on-device array ...
int* idx_end_marker_fila; // on-host array containing index of endpoint of all vortex filaments, for parallel computing
int* dev_idx_end_marker_fila; // on-device array ...

// vel += biot savert induction from segement a-b to position p
__device__ float3 biot_savart_induction(VortexMarker_t a, VortexMarker_t b, float3 p, float3 vel)
{
    float3 ap, bp, ab, ba, ind;
    float dbnrm_ab, dbnrm_ap, dbnrm_bp; // |vector|^2
    float rnrm_ind; // 1/norm(vector) 
    float cos_apab, cos_babp, db_sin_apab;
    float h, scale;

    // if marker a or b is at p, pass
    if ( a.pos[0]==p.x && a.pos[1]==p.y && a.pos[2]==p.z ||
         b.pos[0]==p.x && b.pos[1]==p.y && b.pos[2]==p.z ||
         a.pos[0]==b.pos[0] && a.pos[1]==b.pos[1] && a.pos[2]==b.pos[2]) {
        return vel;
    }

    // get vectors AP, BP and AB & BA  [12 FLOPS]
    //  AP [3 FLOPS]
    /*
    ap.x = p.x - a.pos[0];
    ap.y = p.y - a.pos[1];
    ap.z = p.z - a.pos[2];
    */
    ap.x = __fsub_ru(p.x, a.pos[0]);
    ap.y = __fsub_ru(p.y, a.pos[1]);
    ap.z = __fsub_ru(p.z, a.pos[2]);
    //  BP [3 FLOPS]
    /*
    bp.x = p.x - b.pos[0];
    bp.y = p.y - b.pos[1];
    bp.z = p.z - b.pos[2];
    */
    bp.x = __fsub_ru(p.x, b.pos[0]);
    bp.y = __fsub_ru(p.y, b.pos[1]);
    bp.z = __fsub_ru(p.z, b.pos[2]);
    //  AB [3 FLOPS]
    /*
    ab.x = b.pos[0] - a.pos[0];
    ab.y = b.pos[1] - a.pos[1];
    ab.z = b.pos[2] - a.pos[2];
    */
    ab.x = __fsub_ru(b.pos[0], a.pos[0]);
    ab.y = __fsub_ru(b.pos[1], a.pos[1]);
    ab.z = __fsub_ru(b.pos[2], a.pos[2]);
    //  BA [3 FLOPS]
    /*
    ba.x = a.pos[0] - b.pos[0];
    ba.y = a.pos[1] - b.pos[1];
    ba.z = a.pos[2] - b.pos[2];
    */
    ba.x = __fsub_ru(a.pos[0], b.pos[0]);
    ba.y = __fsub_ru(a.pos[1], b.pos[1]);
    ba.z = __fsub_ru(a.pos[2], b.pos[2]);

    // cos(ap-ab) and cos(ba-bp) [31 FLOPS]
    dbnrm_ab = ab.x*ab.x + ab.y*ab.y + ab.z*ab.z; // [5 FLOPS]
    dbnrm_ap = ap.x*ap.x + ap.y*ap.y + ap.z*ap.z; // [5 FLOPS]
    dbnrm_bp = bp.x*bp.x + bp.y*bp.y + bp.z*bp.z; // [5 FLOPS] 
    cos_apab = (ap.x*ab.x + ap.y*ab.y + ap.z*ab.z)*__frsqrt_rn(dbnrm_ap*dbnrm_ab); // [8 FLOPS]
    cos_babp = (ba.x*bp.x + ba.y*bp.y + ba.z*bp.z)*__frsqrt_rn(dbnrm_ab*dbnrm_bp); // [8 FLOPS]

    // h, perpendicular distance from P to AB [5 FLOPS]
    db_sin_apab = 1-cos_apab*cos_apab; //[2 FLOPS]
    if (db_sin_apab <= 0.0f) // sometimes |cos_apab| will be slightly larger than 1.0f due to computation err
        h = 0.0f;
    else
        h = __fsqrt_ru(dbnrm_ap) * __fsqrt_ru(db_sin_apab); // [3 FLOPS]
if (h != h)
    printf("fuck");

    // strength
    scale = a.Gamma * (h/(0.0001+h*h)) * (cos_apab+cos_babp) / (4*PI);
 
if (scale != scale)
    printf("vel.x = %f, scale = %f, h = %f\n", vel.x, scale, h);

    // get induced velocity
    ind.x = ap.y*bp.z - ap.z*bp.y;
    ind.y = ap.z*bp.x - ap.x*bp.z;
    ind.z = ap.x*bp.y - ap.y*bp.x;
    rnrm_ind = __frsqrt_rn(ind.x*ind.x + ind.y*ind.y + ind.z*ind.z);
    scale = scale * rnrm_ind;
    ind.x = scale * ind.x;
    ind.y = scale * ind.y;
    ind.z = scale * ind.z;

    // add induced velocity [3 FLOPS]
    vel.x += ind.x;
    vel.y += ind.y;
    vel.z += ind.z;

if (vel.x != vel.x)
    printf("vel.x = %f, scale = %f, h = %f\n", vel.x, scale, h);

    return vel;
}

__device__ float3 tile_calculation_vel_markers
(float3 pos, float3 vel, int tile, int tile_size,int row_sgmts, VortexMarker_t* markers, int* index_end, int num_fila, int num_markers)
{
    int i; // the i-th colum marker in this tile
    int idx_m; // the idx_m -th colum marker of whole N markers
    int idx_fila; // index of fila
    int shared_mem_offset = (tile%row_sgmts)*tile_size;

    bool isend; // the marker to be calculated is an end point or not

    extern __shared__ VortexMarker_t tile_markers[];
 
    // if this tile is not the first tile, we will use the last tile's end
    if (tile > 0)
    {
        isend = false;
        for (idx_fila = 0; idx_fila < num_fila; idx_fila++) {
            if (tile*tile_size-1 == index_end[idx_fila]) {
                isend = true;
                break;
            }
        }
        if (isend == false && tile*tile_size < num_markers)
            vel = biot_savart_induction(markers[tile*tile_size-1], tile_markers[shared_mem_offset], pos, vel); 
    }

    for (i = 0; i < tile_size-1; i++)
    {// compule all colum markers in this tile
        idx_m = tile * tile_size + i;
        if (idx_m >= num_markers-1)
            break;
        else
        {
            isend = false;
            // check whether this marker is end point or not
            for (idx_fila = 0; idx_fila < num_fila; idx_fila++) {
                if (idx_m == index_end[idx_fila]) {
                    isend = true;
                    break;
                }
            }
            if (isend == false) // if this marker is not an end point
                vel = biot_savart_induction(tile_markers[shared_mem_offset+i], tile_markers[shared_mem_offset+i+1], pos, vel);
        }
    }

    return vel;
}

/* calculate velocities of markers, running on GPU
Note:
    This routine uses one-dimensional blocks and threads
    row_sgms is the number of segments to parallelly compute a row
    blockDim must equal to tile_size*row_sgmts
    gridDim == (num_markers + tile_size - 1) / tile_size
 */
__global__ void CalculateVelofMarkers(VortexMarker_t* markers, int* idx_end, int num_fila, int num_markers, int tile_size, int row_sgmts)
{
    extern __shared__ VortexMarker_t tile_markers[];

    int idx, i;
    float3 pos; // position of vortex marker to calculate velocity in this thread
    float3 vel = {0.0f, 0.0f, 0.0f}; // velocity of this marker
    int row = threadIdx.x%tile_size + blockIdx.x*tile_size; // get row number (the index of marker to be evaluated)

    // get the marker of which the velocity to be calculated
    if (row < num_markers) {
        pos.x = markers[row].pos[0];
        pos.y = markers[row].pos[1];
        pos.z = markers[row].pos[2];
    }

    // compute each tile
    i = 0;
    while(true) // every thread can enter this loop
    {
        idx = threadIdx.x + blockDim.x*i;
        if (idx < num_markers)
            tile_markers[threadIdx.x] = markers[idx]; // copy markers to shared mem for tile calculation 
        __syncthreads(); // make sure the shared mem has been loaded
        if (row < num_markers)
            vel = tile_calculation_vel_markers(pos, vel, idx/tile_size, tile_size, row_sgmts, markers, idx_end, num_fila, num_markers);
        __syncthreads(); // make sure every thread has done the calc of this tile
        
        if (blockDim.x*(i+1) < num_markers)
            i++;
        else
            break;
    }

    // save the result of this segment to the end of the shared mem
    //  at present the shared memory is of no use for computation, so it can be used to temporarily store the segment results
    if (row < num_markers) {
        tile_markers[threadIdx.x].vel[0] = vel.x;
        tile_markers[threadIdx.x].vel[1] = vel.y;
        tile_markers[threadIdx.x].vel[2] = vel.z;
    }
    __syncthreads(); // make sure the segment results are stored
 
    if (threadIdx.x < tile_size && row < num_markers) {
        // sum the velocities computed by multiple (num of row_sgms) threads
        vel.x = 0.0f; vel.y = 0.0f; vel.z = 0.0f;
        for (i = 0; i < row_sgmts; i++) {
            vel.x += tile_markers[threadIdx.x%tile_size+i*tile_size].vel[0];
            vel.y += tile_markers[threadIdx.x%tile_size+i*tile_size].vel[1];
            vel.z += tile_markers[threadIdx.x%tile_size+i*tile_size].vel[2];
        }
        // Save the result in global memory for the integration step.
        markers[row].vel[0] = vel.x;
        markers[row].vel[1] = vel.y;
        markers[row].vel[2] = vel.z;

        markers[row].pos[0] += vel.x*0.01;
        markers[row].pos[1] += vel.y*0.01;
        markers[row].pos[2] += vel.z*0.01;
    }
}
#if 0
/* calculate new positions of markers, running on GPU */
__global__ void CalculatePosofMarkers(VortexMarker_t* markers, int num_markers)
{
    VortexMarker_t mkr;
    float3 pos, vel;
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if (tid < num_markers) {
        mkr = markers[tid];

        for (int i = 0; i < 3; i++)
            mkr.pos[i] = mkr.vel[i]*0.01;
    }
    __syncthreads();
    if (tid < num_markers)
        markers[tid] = mkr;
}
#endif

/* update all of the rotor wakes in the environment
 * this routine will traverse all of the robot instances
 * and update all of the Lagrangian markers for all robots */
void WakesUpdate(std::vector<Robot*>* robots)
{
    int idx_robot, idx_rotor, idx_blade;
    int num_blade = 0;
    int addr_cp_markers = 0; // index for copy marker states from rotor wake to wake_markers

/* Step 1: update velocity & position of markers */
 
    // Phase 1: collect all vortex markers to a memory, for GPU computing
    //  the markers are placed contiguously, fila to fila
    for(idx_robot = 0; idx_robot < robots->size(); idx_robot++) {
        for (idx_rotor = 0; idx_rotor < robots->at(idx_robot)->wakes.size(); idx_rotor++) {
            for (idx_blade = 0; idx_blade < robots->at(idx_robot)->wakes.at(idx_rotor)->rotor_state.frame.n_blades; idx_blade++) {
                std::copy(robots->at(idx_robot)->wakes.at(idx_rotor)->wake_state[idx_blade]->begin(),
                    robots->at(idx_robot)->wakes.at(idx_rotor)->wake_state[idx_blade]->end(),
                    &wake_markers[addr_cp_markers]);
                addr_cp_markers += robots->at(idx_robot)->wakes.at(idx_rotor)->wake_state[idx_blade]->size();
                idx_end_marker_fila[num_blade] = addr_cp_markers-1; // the address of the last element, hence -1
                num_blade++;
            }
        }
    }// traversed all rotor wakes and got total number of markers

    // Phase 2: copy array wake_markers & idx_wake_markers to GPU's dev_wake_markers
    HANDLE_ERROR( hipMemcpy(dev_wake_markers, wake_markers, 
                addr_cp_markers*sizeof(VortexMarker_t), hipMemcpyHostToDevice) );
    HANDLE_ERROR( hipMemcpy(dev_idx_end_marker_fila, idx_end_marker_fila, 
                num_blade*sizeof(int), hipMemcpyHostToDevice) );
    
    // Phase 3: parallel computing
    //  determine threads per block and blocks number, at present addr_cp_markers contains total num of markers
    int p, q, threads, blocks;
    p = prop.warpSize; // tile size
    q = 2; // number of segments of a row
    threads = std::min(p*q, prop.maxThreadsPerBlock);
    blocks = (addr_cp_markers + p - 1) / p;
    //  Note: here omitted checks for max number of blocks, since in RAO problem the vortex markers
    //        rarely exceeds 65535*threads.

    //  launch gpu computing
    //   the end address of shared memory contains the end element of 'last' tile for 'next' tile computation
    hipError_t err;
    CalculateVelofMarkers<<<blocks, threads, (threads)*sizeof(VortexMarker_t)>>>(dev_wake_markers, dev_idx_end_marker_fila, num_blade, addr_cp_markers, p, q);
    err = hipGetLastError();
    if (err != hipSuccess) 
        printf("Error: %s\n", hipGetErrorString(err));
#if 0
    CalculatePosofMarkers<<<blocks, threads>>>(dev_wake_markers, addr_cp_markers);
    err = hipGetLastError();
    if (err != hipSuccess) 
        printf("Error: %s\n", hipGetErrorString(err));
#endif
    // Phase 4: retrieve data dev_wake_markers from GPU to wake_markers
    HANDLE_ERROR( hipMemcpy(wake_markers, dev_wake_markers,
                addr_cp_markers*sizeof(VortexMarker_t), hipMemcpyDeviceToHost) );

    // Phase 5: GPU computed, distribute the memory to every rotor wake states
    addr_cp_markers = 0;
    for(idx_robot = 0; idx_robot < robots->size(); idx_robot++) {
        for (idx_rotor = 0; idx_rotor < robots->at(idx_robot)->wakes.size(); idx_rotor++) {
            for (idx_blade = 0; idx_blade < robots->at(idx_robot)->wakes.at(idx_rotor)->rotor_state.frame.n_blades; idx_blade++) {
                std::copy(&wake_markers[addr_cp_markers],
                    &wake_markers[addr_cp_markers+robots->at(idx_robot)->wakes.at(idx_rotor)->wake_state[idx_blade]->size()],
                robots->at(idx_robot)->wakes.at(idx_rotor)->wake_state[idx_blade]->data());
                addr_cp_markers += robots->at(idx_robot)->wakes.at(idx_rotor)->wake_state[idx_blade]->size();
            }
        }
    }


/* Step 2: maintain markers of wakes */
    for(idx_robot = 0; idx_robot < robots->size(); idx_robot++)
    {
        for (idx_rotor = 0; idx_rotor < robots->at(idx_robot)->wakes.size(); idx_rotor++) {
            robots->at(idx_robot)->wakes.at(idx_rotor)->maintain();
        }
    }
}

/* init parallelization of wakes computation */
void WakesInit(std::vector<Robot*>* robots)
{
    /* print the properties of all the graphic cards this machine has */ 
    int count; // number of devices
    HANDLE_ERROR( hipGetDeviceCount(&count) );
    for (int i = 0; i < count; i++) {// print out info of all graphic cards
        HANDLE_ERROR( hipGetDeviceProperties(&prop, i) );
        printf("======== Card %d ========\n", i+1);
        printf("Graphic card name: %s\n", prop.name);
        printf("Compute Capability: %d.%d\n", prop.major, prop.minor);
        printf("Total global memory: %ld MByte\n", prop.totalGlobalMem/1024/1024);
        printf("Total constant memoty: %ld kByte\n", prop.totalConstMem/1024);
        printf("Multiprocessor count: %d\n", prop.multiProcessorCount);
        printf("Shared mem per mp: %ld\n", prop.sharedMemPerBlock);
        printf("Registers per mp: %d\n", prop.regsPerBlock);
        printf("Threads in warp: %d\n", prop.warpSize);
        printf("Max threads per block: %d\n", prop.maxThreadsPerBlock);
        printf("Max thread dimensions: (%d, %d, %d)\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
        printf("Max grid dimensions: (%d, %d, %d)\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
        printf("\n");
    }
    if (count > 1) {// multiple graphic cards
        printf("Warning: Multiple graphic cards have been found on this machine. Please modify the function WakeInit in the file src/model/wake.cu to choose the most appropriate card.\n");
        exit(EXIT_FAILURE); // force the user to choose which card to use
    }
    else if (count <= 0) {// no graphic card found
        printf("Error: No graphic cards have been found on this machine. Please run this program on the machine with NVIDIA graphic cards.\n");
        exit(EXIT_FAILURE);
    }

    /* Init GPU computation */
    // Note: the number of rotor wakes & vortex filaments are fixed since simulation starts

    // traverse all rotor wakes and got total max number of markers and fila, for allocating mem
    int max_num_markers = 0;
    int max_num_fila = 0;
    for(int idx_robot = 0; idx_robot < robots->size(); idx_robot++) {// traverse all robots
        for (int idx_rotor = 0; idx_rotor < robots->at(idx_robot)->wakes.size(); idx_rotor++) {// traverse all rotors
            for (int idx_blade = 0; idx_blade < robots->at(idx_robot)->wakes.at(idx_rotor)->rotor_state.frame.n_blades; idx_blade++)
            {// traverse all blades
                max_num_fila++;
                max_num_markers += robots->at(idx_robot)->wakes.at(idx_rotor)->max_markers;
            }
        }
    }

    // allocate a page-locked host memory containing all of the marker states
    HANDLE_ERROR( hipHostAlloc((void**)&wake_markers, 
        max_num_markers*sizeof(*wake_markers), hipHostMallocDefault) );
    // allocate device memory as big as the host's
    HANDLE_ERROR( hipMalloc((void**)&dev_wake_markers, 
        max_num_markers*sizeof(*dev_wake_markers)) );

    // allocate host memory containing the index of wake_markers
    HANDLE_ERROR( hipHostAlloc((void**)&idx_end_marker_fila,
        max_num_fila*sizeof(*idx_end_marker_fila), hipHostMallocDefault) );
    // allocate device memory containing the indexes
    HANDLE_ERROR( hipMalloc((void**)&dev_idx_end_marker_fila,
        max_num_fila*sizeof(*dev_idx_end_marker_fila)) ); 
}

/* close GPU computation */
void WakesFinish(void)
{
    // free device memory
    HANDLE_ERROR( hipFree(dev_idx_end_marker_fila) );
    HANDLE_ERROR( hipFree(dev_wake_markers) );
    // free host memory
    HANDLE_ERROR( hipHostFree(idx_end_marker_fila) );
    HANDLE_ERROR( hipHostFree(wake_markers) );
}

/* End of file wake.cu */
