#include "hip/hip_runtime.h"
/*
 * Wakes in the simulation environment
 *
 * Author: Roice (LUO Bing)
 * Date: 2016-03-08 create this file (RAOS)
 */

#include <stdio.h>
#include "model/robot.h"
#include "model/wake_rotor.h"
#include "model/error_cuda.h"

#define PI 3.14159265358979323846

// storing cuda device properties, for the sheduling of parallel computing
static hipDeviceProp_t prop;

// data
VortexMarker_t* wake_markers; // on-host array containing all of the markers, for convenience mem copying with GPU
VortexMarker_t* dev_wake_markers; // on-device array ...
int* idx_end_marker_fila; // on-host array containing index of endpoint of all vortex filaments, for parallel computing
int* dev_idx_end_marker_fila; // on-device array ...

// vel += biot savert induction from segement a-b to position p
__device__ float3 biot_savart_induction(VortexMarker_t a, VortexMarker_t b, float3 p, float3 vel, int num_markers)
{
    float3 ap, bp, ab, ba, ind;
    float dbnrm_ab, dbnrm_ap, dbnrm_bp; // |vector|^2
    float rnrm_ind; // 1/norm(vector) 
    float cos_apab, cos_babp, db_sin_apab;
    float h, scale;

    // if marker a or b is at p, pass
    if ( a.pos[0]==p.x && a.pos[1]==p.y && a.pos[2]==p.z ||
         b.pos[0]==p.x && b.pos[1]==p.y && b.pos[2]==p.z ||
         a.pos[0]==b.pos[0] && a.pos[1]==b.pos[1] && a.pos[2]==b.pos[2]) {
        return vel;
    }

    // get vectors AP, BP and AB & BA  [12 FLOPS]
    //  AP [3 FLOPS]
    ap.x = p.x - a.pos[0];
    ap.y = p.y - a.pos[1];
    ap.z = p.z - a.pos[2];
    //  BP [3 FLOPS]
    bp.x = p.x - b.pos[0];
    bp.y = p.y - b.pos[1];
    bp.z = p.z - b.pos[2];
    //  AB [3 FLOPS]
    ab.x = b.pos[0] - a.pos[0];
    ab.y = b.pos[1] - a.pos[1];
    ab.z = b.pos[2] - a.pos[2];
    //  BA [3 FLOPS]
    ba.x = a.pos[0] - b.pos[0];
    ba.y = a.pos[1] - b.pos[1];
    ba.z = a.pos[2] - b.pos[2];

    // cos(ap-ab) and cos(ba-bp) [31 FLOPS]
    dbnrm_ab = ab.x*ab.x + ab.y*ab.y + ab.z*ab.z; // [5 FLOPS]
    dbnrm_ap = ap.x*ap.x + ap.y*ap.y + ap.z*ap.z; // [5 FLOPS]
    dbnrm_bp = bp.x*bp.x + bp.y*bp.y + bp.z*bp.z; // [5 FLOPS] 
    cos_apab = (ap.x*ab.x + ap.y*ab.y + ap.z*ab.z)*rsqrtf(dbnrm_ap*dbnrm_ab); // [8 FLOPS]
    cos_babp = (ba.x*bp.x + ba.y*bp.y + ba.z*bp.z)*rsqrtf(dbnrm_ab*dbnrm_bp); // [8 FLOPS]

    // h, perpendicular distance from P to AB [5 FLOPS]
    db_sin_apab = 1-cos_apab*cos_apab; //[2 FLOPS]
    if (db_sin_apab <= 0.0f) // sometimes |cos_apab| will be slightly larger than 1.0f due to computation err
        h = 0.0f;
    else
        h = sqrtf(dbnrm_ap) * sqrtf(db_sin_apab); // [3 FLOPS]
if (h != h)
    printf("fuck");

    // strength
    scale = a.Gamma * (h/(0.0001+h*h)) * (cos_apab+cos_babp) / (4*PI);
 
if (scale != scale)
    printf("vel.x = %f, scale = %f, h = %f\n", vel.x, scale, h);

    // get induced velocity
    ind.x = ap.y*bp.z - ap.z*bp.y;
    ind.y = ap.z*bp.x - ap.x*bp.z;
    ind.z = ap.x*bp.y - ap.y*bp.x;
    rnrm_ind = rsqrtf(ind.x*ind.x + ind.y*ind.y + ind.z*ind.z);
    scale = scale * rnrm_ind;
    ind.x = scale * ind.x;
    ind.y = scale * ind.y;
    ind.z = scale * ind.z;

    // add induced velocity [3 FLOPS]
    vel.x += ind.x;
    vel.y += ind.y;
    vel.z += ind.z;

if (vel.x != vel.x)
    printf("vel.x = %f, scale = %f, h = %f\n", vel.x, scale, h);

    return vel;
}

__device__ float3 tile_calculation_vel_markers
(float3 pos, float3 vel, int tile, int* index_end, int num_fila, int num_markers)
{
    int i; // the i-th colum marker in this tile
    int idx_m; // the idx_m -th colum marker of whole N markers
    int idx_fila; // index of fila

    bool isend; // the marker to be calculated is an end point or not

    extern __shared__ VortexMarker_t tile_markers[];
 
//printf("tile_markers[0].pos[0] = %f, tile_markers[1].pos[0] = %f\n", tile_markers[0].pos[0], tile_markers[1].pos[0]);


    // if this tile is not the first tile, we will use the last tile's end
    if (tile > 0)
    {
        isend = false;
        for (idx_fila = 0; idx_fila < num_fila; idx_fila++) {
            if (tile*blockDim.x-1 == index_end[idx_fila]) {
                isend = true;
                break;
            }
        }
        if (isend == false)
            vel = biot_savart_induction(tile_markers[blockDim.x], tile_markers[0], pos, vel, num_markers); 
    }

    for (i = 0; i < blockDim.x -1; i++)
    {// compule all colum markers in this tile
        idx_m = tile * blockDim.x + i;
        if (idx_m >= num_markers-1)
            break;
        else
        {
            isend = false;
            // check whether this marker is end point or not
            for (idx_fila = 0; idx_fila < num_fila; idx_fila++) {
                if (idx_m == index_end[idx_fila]) {
                    isend = true;
                    break;
                }
            }
            if (isend == false) // if this marker is not an end point
                vel = biot_savart_induction(tile_markers[i], tile_markers[i+1], pos, vel, num_markers); 
        }
    }

    if ((tile+1)*blockDim.x-1 < num_markers)
    {// save the last marker of this tile for the computation of next tile
        tile_markers[blockDim.x] = tile_markers[blockDim.x-1];
    }

    return vel;
}

/* calculate velocities of markers, running on GPU
 */
__global__ void CalculateVelofMarkers(VortexMarker_t* markers, int* idx_end, int num_fila, int num_markers)
{
    extern __shared__ VortexMarker_t tile_markers[];

    int tile, idx;
    float3 pos; // position of vortex marker to calculate velocity in this thread
    float3 vel = {0.0f, 0.0f, 0.0f}; // velocity of this marker
    int tid = threadIdx.x + blockIdx.x * blockDim.x; // get ID of this thread

    // get the marker of which the velocity to be calculated
    if (tid < num_markers) {
        pos.x = markers[tid].pos[0]; // in C++, struct can be directly assigned
        pos.y = markers[tid].pos[1];
        pos.z = markers[tid].pos[2];
    }

    for (tile = 0; tile < gridDim.x; tile++) // every thread can enter this loop
    {
        idx = tile * blockDim.x + threadIdx.x;
        if (idx < num_markers)
            tile_markers[threadIdx.x] = markers[idx]; // copy markers to shared mem for tile calculation 
        __syncthreads(); // make sure the shared mem has been loaded
        if (tid < num_markers)
            vel = tile_calculation_vel_markers(pos, vel, tile, idx_end, num_fila, num_markers);
        __syncthreads(); // make sure every thread has done the calc of this tile
    }

    // Save the result in global memory for the integration step.
    if (tid < num_markers) {
        markers[tid].vel[0] = vel.x;
        markers[tid].vel[1] = vel.y;
        markers[tid].vel[2] = vel.z;

        markers[tid].pos[0] += vel.x*0.01;
        markers[tid].pos[1] += vel.y*0.01;
        markers[tid].pos[2] += vel.z*0.01;
    }
}

/* calculate new positions of markers, running on GPU */
__global__ void CalculatePosofMarkers(VortexMarker_t* markers, int num_markers)
{
    VortexMarker_t mkr;
    float3 pos, vel;
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if (tid < num_markers) {
        mkr = markers[tid];

        for (int i = 0; i < 3; i++)
            mkr.pos[i] = mkr.vel[i]*0.01;
    }
    __syncthreads();
    if (tid < num_markers)
        markers[tid] = mkr;
}


/* update all of the rotor wakes in the environment
 * this routine will traverse all of the robot instances
 * and update all of the Lagrangian markers for all robots */
void WakesUpdate(std::vector<Robot*>* robots)
{
    int idx_robot, idx_rotor, idx_blade;
    int num_blade = 0;
    int addr_cp_markers = 0; // index for copy marker states from rotor wake to wake_markers

/* Step 1: update velocity & position of markers */
 
    // Phase 1: collect all vortex markers to a memory, for GPU computing
    //  the markers are placed contiguously, fila to fila
    for(idx_robot = 0; idx_robot < robots->size(); idx_robot++) {
        for (idx_rotor = 0; idx_rotor < robots->at(idx_robot)->wakes.size(); idx_rotor++) {
            for (idx_blade = 0; idx_blade < robots->at(idx_robot)->wakes.at(idx_rotor)->rotor_state.frame.n_blades; idx_blade++) {
                std::copy(robots->at(idx_robot)->wakes.at(idx_rotor)->wake_state[idx_blade]->begin(),
                    robots->at(idx_robot)->wakes.at(idx_rotor)->wake_state[idx_blade]->end(),
                    &wake_markers[addr_cp_markers]);
                addr_cp_markers += robots->at(idx_robot)->wakes.at(idx_rotor)->wake_state[idx_blade]->size();
                idx_end_marker_fila[num_blade] = addr_cp_markers-1; // the address of the last element, hence -1
                num_blade++;
            }
        }
    }// traversed all rotor wakes and got total number of markers

    // Phase 2: copy array wake_markers & idx_wake_markers to GPU's dev_wake_markers
    HANDLE_ERROR( hipMemcpy(dev_wake_markers, wake_markers, 
                addr_cp_markers*sizeof(VortexMarker_t), hipMemcpyHostToDevice) );
    HANDLE_ERROR( hipMemcpy(dev_idx_end_marker_fila, idx_end_marker_fila, 
                num_blade*sizeof(int), hipMemcpyHostToDevice) );
    
    // Phase 3: parallel computing
    //  determine threads per block and blocks number, at present addr_cp_markers contains total num of markers
    int threads, blocks;
    if (addr_cp_markers < 4*prop.warpSize * prop.multiProcessorCount) // N too small, need to cover as many processors as possible
        threads = std::min(2*prop.warpSize, prop.maxThreadsPerBlock);
    else
        threads = std::min(4*prop.warpSize, prop.maxThreadsPerBlock);
    blocks = (addr_cp_markers + threads - 1) / threads; // make sure there are enough blocks&threads
    //  Note: here omitted checks for max number of blocks, since in RAO problem the vortex markers
    //        rarely exceeds 65535*threads.

    //  launch gpu computing
    //   the end address of shared memory contains the end element of 'last' tile for 'next' tile computation
    hipError_t err;

    CalculateVelofMarkers<<<blocks, threads, (threads+1)*sizeof(VortexMarker_t)>>>(dev_wake_markers, dev_idx_end_marker_fila, num_blade, addr_cp_markers);
    err = hipGetLastError();
    if (err != hipSuccess) 
        printf("Error: %s\n", hipGetErrorString(err));
#if 0
    CalculatePosofMarkers<<<blocks, threads>>>(dev_wake_markers, addr_cp_markers);
    err = hipGetLastError();
    if (err != hipSuccess) 
        printf("Error: %s\n", hipGetErrorString(err));
#endif
    // Phase 4: retrieve data dev_wake_markers from GPU to wake_markers
    HANDLE_ERROR( hipMemcpy(wake_markers, dev_wake_markers,
                addr_cp_markers*sizeof(VortexMarker_t), hipMemcpyDeviceToHost) );

    // Phase 5: GPU computed, distribute the memory to every rotor wake states
    addr_cp_markers = 0;
    for(idx_robot = 0; idx_robot < robots->size(); idx_robot++) {
        for (idx_rotor = 0; idx_rotor < robots->at(idx_robot)->wakes.size(); idx_rotor++) {
            for (idx_blade = 0; idx_blade < robots->at(idx_robot)->wakes.at(idx_rotor)->rotor_state.frame.n_blades; idx_blade++) {
                std::copy(&wake_markers[addr_cp_markers],
                    &wake_markers[addr_cp_markers+robots->at(idx_robot)->wakes.at(idx_rotor)->wake_state[idx_blade]->size()],
                robots->at(idx_robot)->wakes.at(idx_rotor)->wake_state[idx_blade]->data());
                addr_cp_markers += robots->at(idx_robot)->wakes.at(idx_rotor)->wake_state[idx_blade]->size();
            }
        }
    }


/* Step 2: maintain markers of wakes */
    for(idx_robot = 0; idx_robot < robots->size(); idx_robot++)
    {
        for (idx_rotor = 0; idx_rotor < robots->at(idx_robot)->wakes.size(); idx_rotor++) {
            robots->at(idx_robot)->wakes.at(idx_rotor)->maintain();
        }
    }
}

/* init parallelization of wakes computation */
void WakesInit(std::vector<Robot*>* robots)
{
    /* print the properties of all the graphic cards this machine has */ 
    int count; // number of devices
    HANDLE_ERROR( hipGetDeviceCount(&count) );
    for (int i = 0; i < count; i++) {// print out info of all graphic cards
        HANDLE_ERROR( hipGetDeviceProperties(&prop, i) );
        printf("======== Card %d ========\n", i+1);
        printf("Graphic card name: %s\n", prop.name);
        printf("Compute Capability: %d.%d\n", prop.major, prop.minor);
        printf("Total global memory: %ld MByte\n", prop.totalGlobalMem/1024/1024);
        printf("Total constant memoty: %ld kByte\n", prop.totalConstMem/1024);
        printf("Multiprocessor count: %d\n", prop.multiProcessorCount);
        printf("Shared mem per mp: %ld\n", prop.sharedMemPerBlock);
        printf("Registers per mp: %d\n", prop.regsPerBlock);
        printf("Threads in warp: %d\n", prop.warpSize);
        printf("Max threads per block: %d\n", prop.maxThreadsPerBlock);
        printf("Max thread dimensions: (%d, %d, %d)\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
        printf("Max grid dimensions: (%d, %d, %d)\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
        printf("\n");
    }
    if (count > 1) {// multiple graphic cards
        printf("Warning: Multiple graphic cards have been found on this machine. Please modify the function WakeInit in the file src/model/wake.cu to choose the most appropriate card.\n");
        exit(EXIT_FAILURE); // force the user to choose which card to use
    }
    else if (count <= 0) {// no graphic card found
        printf("Error: No graphic cards have been found on this machine. Please run this program on the machine with NVIDIA graphic cards.\n");
        exit(EXIT_FAILURE);
    }

    /* Init GPU computation */
    // Note: the number of rotor wakes & vortex filaments are fixed since simulation starts

    // traverse all rotor wakes and got total max number of markers and fila, for allocating mem
    int max_num_markers = 0;
    int max_num_fila = 0;
    for(int idx_robot = 0; idx_robot < robots->size(); idx_robot++) {// traverse all robots
        for (int idx_rotor = 0; idx_rotor < robots->at(idx_robot)->wakes.size(); idx_rotor++) {// traverse all rotors
            for (int idx_blade = 0; idx_blade < robots->at(idx_robot)->wakes.at(idx_rotor)->rotor_state.frame.n_blades; idx_blade++)
            {// traverse all blades
                max_num_fila++;
                max_num_markers += robots->at(idx_robot)->wakes.at(idx_rotor)->max_markers;
            }
        }
    }

    // allocate a page-locked host memory containing all of the marker states
    HANDLE_ERROR( hipHostAlloc((void**)&wake_markers, 
        max_num_markers*sizeof(*wake_markers), hipHostMallocDefault) );
    // allocate device memory as big as the host's
    HANDLE_ERROR( hipMalloc((void**)&dev_wake_markers, 
        max_num_markers*sizeof(*dev_wake_markers)) );

    // allocate host memory containing the index of wake_markers
    HANDLE_ERROR( hipHostAlloc((void**)&idx_end_marker_fila,
        max_num_fila*sizeof(*idx_end_marker_fila), hipHostMallocDefault) );
    // allocate device memory containing the indexes
    HANDLE_ERROR( hipMalloc((void**)&dev_idx_end_marker_fila,
        max_num_fila*sizeof(*dev_idx_end_marker_fila)) ); 
}

/* close GPU computation */
void WakesFinish(void)
{
    // free device memory
    HANDLE_ERROR( hipFree(dev_idx_end_marker_fila) );
    HANDLE_ERROR( hipFree(dev_wake_markers) );
    // free host memory
    HANDLE_ERROR( hipHostFree(idx_end_marker_fila) );
    HANDLE_ERROR( hipHostFree(wake_markers) );
}

/* End of file wake.cu */
