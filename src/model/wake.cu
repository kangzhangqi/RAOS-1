#include "hip/hip_runtime.h"
/*
 * Wakes in the simulation environment
 *
 * Author: Roice (LUO Bing)
 * Date: 2016-03-08 create this file (RAOS)
 */

#include <stdio.h>
#include "model/robot.h"
#include "model/wake_rotor.h"
#include "model/error_cuda.h"

// storing cuda device properties, for the sheduling of parallel computing
static hipDeviceProp_t prop;

// data
VortexMarker_t* wake_markers; // on-host array containing all of the markers, for convenience mem copying with GPU
VortexMarker_t* dev_wake_markers; // on-device array ...
int* idx_end_marker_fila; // on-host array containing index of endpoint of all vortex filaments, for parallel computing
int* dev_idx_end_marker_fila; // on-device array ...

__device__ float3 tile_calculation_vel_markers
(VortexMarker_t vtx_mkr, float3 vel, int tile, int* index_end, int num_fila)
{
    int i; // the i-th colum marker in this tile
    int idx_m; // the idx_m -th colum marker of whole N markers
    int idx_fila; // index of fila

    bool isend; // the marker to be calculated is an end point or not

    static VortexMarker_t prev_tile_end; // the last marker of previous tile
    static int idx_pre_tile_end; // the index of ...

    extern __shared__ VortexMarker_t tile_markers[threadsPerBlock];
   
    // if this tile is not the first tile, we will use the last tile's end
    if (tile != 0)
    {
        isend = false;
        for (idx_fila = 0; idx_fila < num_fila; idx_fila++) {
            if (idx_pre_tile_end == index_end[idx_fila]) {
                isend = true;
                break;
            }
        }
        if (isend == false)
            biot_savart_induction(prev_tile_end, tile_markers[0], ...);
    }

    for (i = 0; i < blockDim.x -1; i++)
    {// compule all colum markers in this tile
        idx_m = tile * blockDim.x + i;
        if (idx_m >= num_markers)
            break;
        else
        {
            isend = false;
            // check whether this marker is end point or not
            for (idx_fila = 0; idx_fila < num_fila; idx_fila++) {
                if (idx_m == index_end[idx_fila]) {
                    isend = true;
                    break;
                }
            }
            if (isend == false) // if this marker is not an end point
                biot_savart_induction();

            if (i == blockDim.x-2)
            {// save the last marker of this tile for the computation of next tile
                prev_tile_end = tile_markers[blockDim.x-1];
                idx_pre_tile_end = idx_m+1;
            }
        }
    }
}

/* calculate velocities of markers, running on GPU
 */
__global__ void CalculateVelofMarkers(VortexMarker_t* markers, int* idx_end, int num_fila)
{
    extern __shared__ VortexMarker_t tile_markers[threadsPerBlock];

    int i = 0, tile = 0, idx;
    VortexMarker_t vtx_mkr; // vortex marker to calculate velocity in this thread
    float3 vel = {0.0f, 0.0f, 0.0f};
    int tid = threadIdx.x + blockIdx.x * blockDim.x; // get ID of this thread

    // get the marker of which the velocity to be calculated
    if (tid < num_markers)
        vtx_mkr = markers[tid]; // in C++, struct can be directly assigned

    while (1) // every thread can enter this loop
    {
        idx = tile * blockDim.x + threadIdx.x;
        if (idx < num_markers)
            tile_markers[threadIdx.x] = markers[idx]; // copy markers to shared mem for tile calculation
        __syncthreads(); // make sure the shared mem has been loaded
        if (idx < num_markers)
            tile_calculation_vel_markers(vtx_mkr, vel, tile, idx_end, num_fila);

    }



    // calculate the velocity of this marker
    for (i = 0, tile = 0; i < num_markers; i += blockDim.x, tile++) 
    {
         
__syncthreads();
acc = tile_calculation(myPosition, acc);
__syncthreads();
}
// Save the result in global memory for the integration step.
float4 acc4 = {acc.x, acc.y, acc.z, 0.0f};
globalA[gtid] = acc4;

}


/* update all of the rotor wakes in the environment
 * this routine will traverse all of the robot instances
 * and update all of the Lagrangian markers for all robots */
void WakesUpdate(std::vector<Robot*>* robots)
{
    int idx_robot, idx_rotor, idx_blade;
    int num_blade = 0;
    int addr_cp_markers = 0; // index for copy marker states from rotor wake to wake_markers

/* Step 1: update velocity & position of markers */
 
    // Phase 1: collect all vortex markers to a memory, for GPU computing
    //  the markers are placed contiguously, fila to fila
    for(idx_robot = 0; idx_robot < robots->size(); idx_robot++) {
        for (idx_rotor = 0; idx_rotor < robots->at(idx_robot)->wakes.size(); idx_rotor++) {
            for (idx_blade = 0; idx_blade < robots->at(idx_robot)->wakes.at(idx_rotor)->rotor_state.frame.n_blades; idx_blade++) {
                std::copy(robots->at(idx_robot)->wakes.at(idx_rotor)->wake_state[idx_blade]->begin(),
                    robots->at(idx_robot)->wakes.at(idx_rotor)->wake_state[idx_blade]->end(),
                    &wake_markers[addr_cp_markers]);
                addr_cp_markers += robots->at(idx_robot)->wakes.at(idx_rotor)->wake_state[idx_blade]->size();
                idx_end_marker_fila[num_blade] = addr_cp_markers-1; // the address of the last element, hence -1
                num_blade++;
            }
        }
    }// traversed all rotor wakes and got total number of markers

    // Phase 2: copy array wake_markers & idx_wake_markers to GPU's dev_wake_markers
    HANDLE_ERROR( hipMemcpy(dev_wake_markers, wake_markers, 
                addr_cp_markers*sizeof(VortexMarker_t), hipMemcpyHostToDevice) );
    HANDLE_ERROR( hipMemcpy(dev_idx_end_marker_fila, idx_end_marker_fila, 
                num_blade*sizeof(int), hipMemcpyHostToDevice) );
    
    // Phase 3: parallel computing
    //  determine threads per block and blocks number, at present addr_cp_markers contains total num of markers
    int threads, blocks;
    if (addr_cp_markers < 2*prop.warpSize * prop.multiProcessorCount) // N too small, need to cover as many processors as possible
        threads = imin(prop.warpSize, prop.maxThreadsPerBlock);
    else
        threads = imin(2*prop.warpSize, prop.maxThreadsPerBlock);
    blocks = (addr_cp_markers + threads - 1) / threads; // make sure there are enough blocks&threads
    //  Note: here omitted checks for max number of blocks, since in RAO problem the vortex markers
    //        rarely exceeds 65535*threads.

    //  launch gpu computing
    CalculateVelofMarkers<<<blocks, threads>>>(dev_wake_markers, dev_idx_end_marker_fila, num_blade);


    // Phase 4: retrieve data dev_wake_markers from GPU to wake_markers
    HANDLE_ERROR( hipMemcpy(wake_markers, dev_wake_markers,
                addr_cp_markers*sizeof(VortexMarker_t), hipMemcpyDeviceToHost) );

    // Phase 5: GPU computed, distribute the memory to every rotor wake states
    addr_cp_markers = 0;
    for(idx_robot = 0; idx_robot < robots->size(); idx_robot++) {
        for (idx_rotor = 0; idx_rotor < robots->at(idx_robot)->wakes.size(); idx_rotor++) {
            for (idx_blade = 0; idx_blade < robots->at(idx_robot)->wakes.at(idx_rotor)->rotor_state.frame.n_blades; idx_blade++) {
                std::copy(&wake_markers[addr_cp_markers],
                    &wake_markers[addr_cp_markers+robots->at(idx_robot)->wakes.at(idx_rotor)->wake_state[idx_blade]->size()],
                robots->at(idx_robot)->wakes.at(idx_rotor)->wake_state[idx_blade]->data());
                addr_cp_markers += robots->at(idx_robot)->wakes.at(idx_rotor)->wake_state[idx_blade]->size();
            }
        }
    }


/* Step 2: maintain markers of wakes */
    for(idx_robot = 0; idx_robot < robots->size(); idx_robot++)
    {
        for (idx_rotor = 0; idx_rotor < robots->at(idx_robot)->wakes.size(); idx_rotor++) {
            robots->at(idx_robot)->wakes[idx_rotor]->maintain();
        }
    }
}

/* init parallelization of wakes computation */
void WakesInit(std::vector<Robot*>* robots)
{
    /* print the properties of all the graphic cards this machine has */ 
    int count; // number of devices
    HANDLE_ERROR( hipGetDeviceCount(&count) );
    for (int i = 0; i < count; i++) {// print out info of all graphic cards
        HANDLE_ERROR( hipGetDeviceProperties(&prop, i) );
        printf("======== Card %d ========\n", i+1);
        printf("Graphic card name: %s\n", prop.name);
        printf("Compute Capability: %d.%d\n", prop.major, prop.minor);
        printf("Total global memory: %ld MByte\n", prop.totalGlobalMem/1024/1024);
        printf("Total constant memoty: %ld kByte\n", prop.totalConstMem/1024);
        printf("Multiprocessor count: %d\n", prop.multiProcessorCount);
        printf("Shared mem per mp: %ld\n", prop.sharedMemPerBlock);
        printf("Registers per mp: %d\n", prop.regsPerBlock);
        printf("Threads in warp: %d\n", prop.warpSize);
        printf("Max threads per block: %d\n", prop.maxThreadsPerBlock);
        printf("Max thread dimensions: (%d, %d, %d)\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
        printf("Max grid dimensions: (%d, %d, %d)\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
        printf("\n");
    }
    if (count > 1) {// multiple graphic cards
        printf("Warning: Multiple graphic cards have been found on this machine. Please modify the function WakeInit in the file src/model/wake.cu to choose the most appropriate card.\n");
        exit(EXIT_FAILURE); // force the user to choose which card to use
    }
    else if (count <= 0) {// no graphic card found
        printf("Error: No graphic cards have been found on this machine. Please run this program on the machine with NVIDIA graphic cards.\n");
        exit(EXIT_FAILURE);
    }

    /* Init GPU computation */
    // Note: the number of rotor wakes & vortex filaments are fixed since simulation starts

    // traverse all rotor wakes and got total max number of markers and fila, for allocating mem
    int max_num_markers = 0;
    int max_num_fila = 0;
    for(int idx_robot = 0; idx_robot < robots->size(); idx_robot++) {// traverse all robots
        for (int idx_rotor = 0; idx_rotor < robots->at(idx_robot)->wakes.size(); idx_rotor++) {// traverse all rotors
            for (int idx_blade = 0; idx_blade < robots->at(idx_robot)->wakes.at(idx_rotor)->rotor_state.frame.n_blades; idx_blade++)
            {// traverse all blades
                max_num_fila++;
                max_num_markers += robots->at(idx_robot)->wakes.at(idx_rotor)->max_markers;
            }
        }
    }

    // allocate a page-locked host memory containing all of the marker states
    HANDLE_ERROR( hipHostAlloc((void**)&wake_markers, 
        max_num_markers*sizeof(*wake_markers), hipHostMallocDefault) );
    // allocate device memory as big as the host's
    HANDLE_ERROR( hipMalloc((void**)&dev_wake_markers, 
        max_num_markers*sizeof(*dev_wake_markers)) );

    // allocate host memory containing the index of wake_markers
    HANDLE_ERROR( hipHostAlloc((void**)&idx_end_marker_fila,
        max_num_fila*sizeof(*idx_end_marker_fila), hipHostMallocDefault) );
    // allocate device memory containing the indexes
    HANDLE_ERROR( hipMalloc((void**)&dev_idx_end_marker_fila,
        max_num_fila*sizeof(*dev_idx_end_marker_fila)) ); 
}

/* close GPU computation */
void WakesFinish(void)
{
    // free device memory
    HANDLE_ERROR( hipFree(dev_idx_end_marker_fila) );
    HANDLE_ERROR( hipFree(dev_wake_markers) );
    // free host memory
    HANDLE_ERROR( hipHostFree(idx_end_marker_fila) );
    HANDLE_ERROR( hipHostFree(wake_markers) );
}

/* End of file wake.cu */
