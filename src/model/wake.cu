#include "hip/hip_runtime.h"
/*
 * Wakes in the simulation environment
 *
 * Author: Roice (LUO Bing)
 * Date: 2016-03-08 create this file (RAOS)
 */

#include <stdio.h>
#include "model/robot.h"
#include "model/wake_rotor.h"
#include "model/error_cuda.h"

VortexMarker_t* wake_markers; // on-host array containing all of the markers, for convenience mem copying with GPU
VortexMarker_t* dev_wake_markers; // on-device array ...

/* update all of the rotor wakes in the environment
 * this routine will traverse all of the robot instances
 * and update all of the Lagrangian markers for all robots */
void WakesUpdate(std::vector<Robot*>* robots)
{
    int idx_robot, idx_rotor;
    int idx_wake_markers = 0; // index for copy marker states from rotor wake to wake_markers

/* Step 1: update velocity & position of markers */
 
    // Phase 1: collect all rotor wake states to a memory, for GPU computing
    for(idx_robot = 0; idx_robot < robots->size(); idx_robot++) {
        for (idx_rotor = 0; idx_rotor < robots->at(idx_robot)->wakes.size(); idx_rotor++) {
            std::copy(robots->at(idx_robot)->wakes[idx_rotor]->wake_state.begin(),
                    robots->at(idx_robot)->wakes[idx_rotor]->wake_state.end(),
                    &wake_markers[idx_wake_markers]);
            idx_wake_markers += robots->at(idx_robot)->wakes[idx_rotor]->wake_state.size();
        }
    }// traversed all rotor wakes and got total number of markers

    // Phase 2: copy array wake_markers to GPU's dev_wake_markers
    HANDLE_ERROR( hipMemcpy(dev_wake_markers, wake_markers, 
                idx_wake_markers*sizeof(VortexMarker_t), hipMemcpyHostToDevice) );

    // Phase 3: parallel computing


    // Phase 4: retrieve data dev_wake_markers from GPU to wake_markers
    HANDLE_ERROR( hipMemcpy(wake_markers, dev_wake_markers,
                idx_wake_markers*sizeof(VortexMarker_t), hipMemcpyDeviceToHost) );

    // Phase 5: GPU computed, distribute the memory to every rotor wake states
    idx_wake_markers = 0;
    for(idx_robot = 0; idx_robot < robots->size(); idx_robot++) {
        for (idx_rotor = 0; idx_rotor < robots->at(idx_robot)->wakes.size(); idx_rotor++) {
            std::copy(&wake_markers[idx_wake_markers],
                &wake_markers[idx_wake_markers+robots->at(idx_robot)->wakes[idx_rotor]->wake_state.size()], robots->at(idx_robot)->wakes[idx_rotor]->wake_state.data());
            idx_wake_markers += robots->at(idx_robot)->wakes[idx_rotor]->wake_state.size();
        }
    }


/* Step 2: maintain markers of wakes */
    for(idx_robot = 0; idx_robot < robots->size(); idx_robot++)
    {
        for (idx_rotor = 0; idx_rotor < robots->at(idx_robot)->wakes.size(); idx_rotor++) {
            robots->at(idx_robot)->wakes[idx_rotor]->maintain();
        }
    }
}

/* init parallelization of wakes computation */
void WakesInit(std::vector<Robot*>* robots)
{
    /* print the properties of all the graphic cards this machine has */
    hipDeviceProp_t prop;
    int count; // number of devices
    HANDLE_ERROR( hipGetDeviceCount(&count) );
    for (int i = 0; i < count; i++) {// print out info of all graphic cards
        HANDLE_ERROR( hipGetDeviceProperties(&prop, i) );
        printf("======== Card %d ========\n", i+1);
        printf("Graphic card name: %s\n", prop.name);
        printf("Compute Capability: %d.%d\n", prop.major, prop.minor);
        printf("Total global memory: %ld MByte\n", prop.totalGlobalMem/1024/1024);
        printf("Total constant memoty: %ld kByte\n", prop.totalConstMem/1024);
        printf("\n");
    }
    if (count > 1) {// multiple graphic cards
        printf("Warning: Multiple graphic cards have been found on this machine. Please modify the function WakeInit in the file src/model/wake.cu to choose the most appropriate card.\n");
        exit(EXIT_FAILURE); // force the user to choose which card to use
    }
    else if (count <= 0) {// no graphic card found
        printf("Error: No graphic cards have been found on this machine. Please run this program on the machine with NVIDIA graphic cards.\n");
        exit(EXIT_FAILURE);
    }

    /* Init GPU computation */

    // allocate a page-locked host memory containing all of the marker states
    int max_num_markers = 0;
    for(int idx_robot = 0; idx_robot < robots->size(); idx_robot++) {
        for (int idx_rotor = 0; idx_rotor < robots->at(idx_robot)->wakes.size(); idx_rotor++) {
            max_num_markers += robots->at(idx_robot)->wakes[idx_rotor]->max_markers;
        }
    }// traversed all rotor wakes and got total max number of markers
    HANDLE_ERROR( hipHostAlloc((void**)&wake_markers, 
        max_num_markers*sizeof(*wake_markers), hipHostMallocDefault) );

    // allocate device memory as big as the host's
    HANDLE_ERROR( hipMalloc((void**)&dev_wake_markers, max_num_markers*sizeof(*dev_wake_markers)) );
}

/* close GPU computation */
void WakesFinish(void)
{
    // free device memory
    HANDLE_ERROR( hipFree(dev_wake_markers) );
    // free host memory
    HANDLE_ERROR( hipHostFree(wake_markers) );
}

/* End of file wake.cu */
